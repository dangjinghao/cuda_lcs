// it is used to avoid VS intellisense warning for calling cuda kernel_add function 
#ifdef __INTELLISENSE__
#define CUDA_KERNEL_ARGS(...)
#define __CUDACC__
#else
#define CUDA_KERNEL_ARGS(...) <<< __VA_ARGS__ >>>
#endif


#include <iostream>
#include <iterator>
#include <algorithm>
#include <hip/hip_runtime.h> 

#include <assert.h>


constexpr auto required_block_dim = 20;

constexpr auto blockDIM = std::min<int>(required_block_dim, 16);

struct dp_table {
	// mark the table content as `volatile`, the volatile load/store rules will be applied
	// this situation caused a amazing problem: printing in cuda or debugging with break points can promise this program caculate out the result as right situation.
	volatile int* dev_DP_table_ptr;
	const int weight, height;
	const char* dev_str1;
	const char* dev_str2;

	__device__ bool update_when_available(int i,int j) {
		int skip_a = get_value(i - 1, j);
		int skip_b = get_value(i, j - 1);
		int skip_middle = get_value(i - 1, j - 1);
		//printf("[%d,%d]: %d %d %d\n", i, j, skip_a,skip_a,skip_middle);

		//no result
		if (skip_a == -1 || skip_b == -1 || skip_middle == -1) {
			return false;
		}

		int take_both = skip_middle + (dev_str1[i] == dev_str2[j]);
		//update
		set_value(i,j,max(take_both, max(skip_a, skip_b)));
		return true;
	}
	__device__ int get_value(int i, int j) {
		if(i < 0 || j < 0) return 0;
		assert(j < weight);
		assert(i < height);
		return dev_DP_table_ptr[i * weight + j];
	}
	__device__ void set_value(int i, int j, int v) {
		dev_DP_table_ptr[i * weight + j] = v;

	}
};


__global__ void lcs_kernel(dp_table dp) {
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < dp.height && j < dp.weight) {

		//spinning before updated
		while (true) {
			if (dp.update_when_available(i, j)) break;
		}

	}

	
}

auto main() -> int {

	unsigned int str1_len,str2_len;
	std::cin >> str1_len >> str2_len;

	char* str1_arr_0copy;
	hipHostAlloc( (void**)&str1_arr_0copy,sizeof(char)*str1_len, hipHostMallocMapped);

	char* str2_arr_0copy;
	hipHostAlloc((void**)&str2_arr_0copy, sizeof(char) * str2_len, hipHostMallocMapped);


	// When reading characters, istream_iterator skips whitespace by default 
	std::copy_n(std::istream_iterator<char>(std::cin), str1_len, str1_arr_0copy);

	std::copy_n(std::istream_iterator<char>(std::cin), str2_len, str2_arr_0copy);


	// create DP table
	int* dev_DP_table;
	hipMalloc((void**)&dev_DP_table, str1_len * str2_len * sizeof(int));
	//fill the table with -1 to mark the unit is not result
	hipMemset((void*)dev_DP_table, -1, str1_len * str2_len * sizeof(int));


	dim3 threads{ blockDIM,blockDIM };
	// those not aligned weight string may cannot caculate out every value,so we need  +1 to start more thread
	dim3 blocks{ str1_len / blockDIM + 1,str2_len / blockDIM + 1 };
	char*dev_str1_arr,*dev_str2_arr;
	hipHostGetDevicePointer(&dev_str1_arr, str1_arr_0copy, 0);
	hipHostGetDevicePointer(&dev_str2_arr, str2_arr_0copy, 0);

	dp_table dp{dev_DP_table, str1_len,str2_len , dev_str1_arr ,dev_str2_arr};


	lcs_kernel CUDA_KERNEL_ARGS(blocks, threads)(dp);

	int output;
	//copy the result in last unit from device memory to host memory
	hipMemcpy((void*)&output, (void*)(dp.dev_DP_table_ptr + str1_len * str2_len - 1), 1 * sizeof(int), hipMemcpyDeviceToHost);

	std::cout << output<<std::endl;
	hipFree(dev_DP_table);
	hipHostFree(str1_arr_0copy);
	hipHostFree(str2_arr_0copy);
	return 0;
}